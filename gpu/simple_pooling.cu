
#include <hip/hip_runtime.h>
#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>
using namespace std;

__global__ void pooling(int *pixels, int *convolution, int width, int height, int use_max) {
	int convolution_idx = blockIdx.x * blockDim.x + threadIdx.x;
	// int top_left = (blockIdx.x * width) + (threadIdx.x);
	// int top_right = top_left + 1;
	// int bot_left = top_left + width;
	// int bot_right = bot_left + 1;

	if (use_max == 0) { // use mean
		convolution[convolution_idx] =
				__double2int_rn(
						(pixels[((blockIdx.x * width) + (threadIdx.x))]
								+ pixels[((blockIdx.x * width) + (threadIdx.x))
										+ 1]
								+ pixels[(((blockIdx.x * width) + (threadIdx.x))
										+ width)]
								+ pixels[(((blockIdx.x * width) + (threadIdx.x))
										+ width) + 1]

						) / 4.0);
	} else {
		int max = -9999;
		int *window = new int[4] { pixels[((blockIdx.x * width) + (threadIdx.x))],
					pixels[((blockIdx.x * width) + (threadIdx.x)) + 1],
					pixels[(((blockIdx.x * width) + (threadIdx.x)) + width)],
					pixels[(((blockIdx.x * width) + (threadIdx.x)) + width) + 1] };

		for (int idx = 0; idx < 4; idx++) {
			if (window[idx] > max) {
				max = window[idx];
			}
		}
		convolution[convolution_idx] = max;
	}
}

extern "C" int *pooling(int *pixels, const int width, const int height, const int recurse_cnt, const int use_max) {
	int w = width;
	int h = height;
	int size = w * h;
	int convo_size = (w - 1) * (h - 1);
	int *d_pixels;
	int *d_convolution;

	hipMalloc(&d_pixels, size * sizeof(int));
	hipMalloc(&d_convolution, convo_size * sizeof(int));

	hipMemcpy(d_pixels, pixels, size * sizeof(int), hipMemcpyHostToDevice);

	for (int recurse_idx = 0; recurse_idx < recurse_cnt; recurse_idx++) {
		size = w * h;
		convo_size = (w - 1) * (h - 1);

		if (recurse_idx % 2 == 0) {
			pooling<<< (h - 1), (w - 1) >>>(d_pixels, d_convolution, w, h, use_max);
		} else {
			pooling<<< (h - 1), (w - 1) >>>(d_convolution, d_pixels, w, h, use_max);
		}
		w--;
		h--;
	}

	int *convolution = new int[convo_size];
	if (recurse_cnt % 2 == 0) {
		hipMemcpy(convolution, d_pixels, convo_size * sizeof(int),
				hipMemcpyDeviceToHost);
	} else {
		hipMemcpy(convolution, d_convolution, convo_size * sizeof(int),
				hipMemcpyDeviceToHost);
	}

	hipFree(d_pixels);
	hipFree(d_convolution);
	return convolution;
}

int *init_array(const int size) {
	int * array = new int[size];
	for (int i = 0; i < size; i++) {
		array[i] = rand() % 100;
	}
	return array;
}

//  nvcc --ptxas-options=-v --compiler-options '-fPIC' -o gpu.so --shared  simple_poolings.cu
int main(void) {
	const int width = 4;
	const int height = 4;
	const int recurse_cnt = 3;
	const int use_max = 0;
	//int *pixels = new int[16] { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13,14, 15 };
	int * pixels = init_array(1000000);
	int *convolution = pooling(pixels, width, height, recurse_cnt, use_max);
	cout << "\n\nend\n";
	for (int i = 0; i < ((width - recurse_cnt) * (height - recurse_cnt)); i++) {
		std::cout << i << " : " << convolution[i] << " \n";
	}
	return 0;
}
