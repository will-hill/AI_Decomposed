#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <numeric>
#include <algorithm>

__global__ void pooling(int *pixels, int *convolution, int width, int height, int use_max) {
    int i = threadIdx.x;

    if ((i != 0) && ((i + 1) % width == 0)) {
        return; // too far to right
    }

    // grab 2x2 window, calculate max or mean and add to convolution
    int *window = new int[4]{pixels[i], pixels[(i + 1)], pixels[(i + width)], pixels[(i + 1 + width)]};
    if (use_max == 0) { // use mean

        convolution[i] = (pixels[i] + pixels[(i + 1)] + pixels[(i + width)] + pixels[(i + 1 + width)]) / 4;

    } else {

        int max = -9999;

        for (int idx = 0; idx < 4; idx++) {
            if (window[idx] > max) {
                max = window[idx];
            }
        }

        convolution[i] = max;
    }
}

extern "C" int *pooling(int *pixels, const int width, const int height, const int use_max) {


    const int SIZE = width * height;
    const int CONVO_SIZE = (width - 1) * (height - 1);

    int *d_pixels;
    int *d_convolution;
    int *convolution = new int[CONVO_SIZE];

    hipMallocManaged(&d_pixels, SIZE * sizeof(int));
    hipMallocManaged(&d_convolution, CONVO_SIZE * sizeof(int));

    hipMemcpy(d_pixels, pixels, SIZE * sizeof(int), hipMemcpyHostToDevice);

    // no need to calculate last row
    pooling << < 1, (SIZE - height - 1) >> > (d_pixels, d_convolution, width, height, use_max);

    hipDeviceSynchronize();

    hipMemcpy(convolution, d_convolution, CONVO_SIZE * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_pixels);
    hipFree(d_convolution);

    return convolution;
}

int main() {
    const int width = 4;
    const int height = 4;
    const int use_max = 0;

    int *pixels = new int[16]{0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 12, 13, 14, 15};

    int *convolution = pooling(pixels, width, height, use_max);

    for (int i = 0 ; i < 9 ; i++){
        std::cout << i << " : " << pixels[i] << "," << pixels[(i + 1)] << "," <<  pixels[(i + width)] << "," <<  pixels[(i + 1 + width)] << " --> " << convolution[i] << " \n";
    }
    return 0;
}

