#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

__global__ void kernel(int *data) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    doSomeStuff(idx, data, ...);
}


int main(){
    int *data;
    data = 17;
    hipMallocManaged(&data, N * sizeof(int));
    // initialize data on the CPU
    kernel<<<grid, block>>>(data);
}
