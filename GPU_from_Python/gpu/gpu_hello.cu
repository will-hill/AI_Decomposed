#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <numeric>
#include <algorithm>

__global__ void pooling ( int *pixels, int *convolution, int width, int height, int use_max ) {

    int convolution_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int top_left = (blockIdx.x * width) + (threadIdx.x);
    int top_right = top_left + 1;
    int bot_left = top_left + width;
    int bot_right = bot_left + 1;

    // grab 2x2 window, calculate max or mean and add to convolution
    int *window = new int[4]{
            pixels[top_left], pixels[top_right],
            pixels[bot_left], pixels[bot_right]
    };

    if (use_max == 0) { // use mean
        double mean = (pixels[top_left] + pixels[top_right] + pixels[bot_left] + pixels[bot_right]) / 4.0;
        int mean_int = __double2int_rn(mean);

        /*printf("%d: window: %d  %d  %d  %d        %d %d %d %d : %f : %d \n",
               convolution_idx, top_left, top_right, bot_left, bot_right,

               pixels[top_left], pixels[top_right],
               pixels[bot_left], pixels[bot_right],

               mean,
               mean_int);*/


        convolution[convolution_idx] = mean_int;
    } else {

        int max = -9999;
        for (int idx = 0; idx < 4; idx++) {
            if (window[idx] > max) {
                max = window[idx];
            }
        }
        convolution[convolution_idx] = max;
    }
}

extern "C" int test(int *pixels) {
    return -999;
}

extern "C" int *pooling(int *pixels, const int width, const int height, const int use_max) {


    const int SIZE = width * height;
    const int CONVO_SIZE = (width - 1) * (height - 1);

    int *d_pixels;
    int *d_convolution;
    int *convolution = new int[CONVO_SIZE];

    hipMallocManaged(&d_pixels, SIZE * sizeof(int));
    hipMallocManaged(&d_convolution, CONVO_SIZE * sizeof(int));

    hipMemcpy(d_pixels, pixels, SIZE * sizeof(int), hipMemcpyHostToDevice);

    // no need to calculate last row
    pooling <<< (height - 1), (width - 1) >>> (d_pixels, d_convolution, width, height, use_max);

    hipDeviceSynchronize();

    hipMemcpy(convolution, d_convolution, CONVO_SIZE * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_pixels);
    hipFree(d_convolution);

    return convolution;
}

//  nvcc --ptxas-options=-v --compiler-options '-fPIC' -o gpu.so --shared  gpu_hello.cu
int main() {
    const int width = 4;
    const int height = 4;
    const int use_max = 0;

    int *pixels = new int[16]{0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15};

    int *convolution = pooling(pixels, width, height, use_max);

    for (int i = 0; i < 9; i++) {
        std::cout << i << " : " << convolution[i] << " \n";
    }
    return 0;
}

