#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <iostream>


#include <iostream>
#include <vector>
#include <algorithm>
#include <numeric>
#include <tgmath.h>
#include <array>

std::vector<int> pool_2x2_kernel(std::vector<int> pixels, const int width, const int height, const int funct) {
    // total pixels to convolve
    int total_pixels = width * height;
    // to hold convolution
    std::vector<int> convolution;
    // slide kernel over pixels
    for (int i = 0; i < total_pixels; i++) {
        // test if done
        if (i >= (width * (height - 1))) {
            break;
        }
        if (i != 0 and (i + 1) % width == 0) {
            continue;
        }
        // grab 2x2 window, calculate max or mean and add to convolution
        std::vector<int> window{pixels[i], pixels[(i + 1)], pixels[(i + width)], pixels[(i + 1 + width)]};

        double y = 255;
        if (funct == 0) {
            double sum = std::accumulate(window.begin(), window.end(), 0.0);
            y = sum / window.size();
        } else {
            y = *max_element(window.begin(), window.end());
        }
        convolution.push_back(std::nearbyint(y));
    }
    return convolution;
}

extern "C" int *
recurse_convolution(const int *pixels, const int width, const int height, const int recurse_cnt, const int use_max) {
    int w = width, h = height, pixel_count = h * w;
    std::vector<int> convolution(pixel_count);
    for (int i = 0; i < w * h; i++) {
        convolution[i] = pixels[i];
    }
    for (int i = 0; i < recurse_cnt; i++) {
        convolution = pool_2x2_kernel(convolution, w, h, use_max);
        w--;
        h--;
    }

    int *convo_array = new int[convolution.size()];
    std::copy(convolution.begin(), convolution.end(), convo_array);
    return convo_array;
}

int CPU_main(int argc, char *argv[]) {
    (void) argc;

    int mat[] = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15};
    int width = 4;
    int recurse_cnt = 2;

    int mat_size = sizeof(mat) / sizeof(mat[0]);
    int height = mat_size / width;
    int use_max = 0;

    int *output = recurse_convolution(mat, width, height, recurse_cnt, use_max);

    int ret_w = width - recurse_cnt;
    int ret_h = height - recurse_cnt;
    int array_size = ret_h * ret_w;
    std::cout << "---" << std::endl;
    for (int i = 0; i < array_size; i++) {
        std::cout << output[i] << " ";
    }
    //max_2x2_kernel(asdf)
    return 0;
}

using namespace std;


__global__ void vector_add(float *out, float *a, float *b, int n) {
    for (int i = 0; i < n; i++) {
        out[i] = a[i] + b[i];
    }
}

const int N = 16;
const int blocksize = 16;

__global__
void hello(char *a, int *b)
{
    a[threadIdx.x] += b[threadIdx.x];
}

int GPU_DIRECT_main() {

    char a[N] = "Hello \0\0\0\0\0\0";
    int b[N] = {15, 10, 6, 0, -11, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

    char *ad;
    int *bd;
    const int csize = N*sizeof(char);
    const int isize = N*sizeof(int);

    printf("%s", a);

    hipMalloc( (void**)&ad, csize );
    hipMalloc( (void**)&bd, isize );

    hipMemcpy( ad, a, csize, hipMemcpyHostToDevice );
    hipMemcpy( bd, b, isize, hipMemcpyHostToDevice );

    dim3 dimBlock( blocksize, 1 );
    dim3 dimGrid( 1, 1 );

    hello<<<dimGrid, dimBlock>>>(ad, bd);

    hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost );


    hipFree( ad );
    hipFree( bd );

    printf("%s\n", a);
    return 0;
}

int blah() {
    std::cout << "start\n";
    float *a, *b, *out;
    float *d_a;

    // Allocate memory
    a = (float *) malloc(sizeof(float) * N);
    b = (float *) malloc(sizeof(float) * N);
    out = (float *) malloc(sizeof(float) * N);

    // Initialize array
    for (int i = 0; i < N; i++) {
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    for (int i = 0; i < N; i++) {
        std::cout << a[i] << " ";
    }
    std::cout << endl;

    for (int i = 0; i < N; i++) {
        std::cout << b[i] << " ";
    }
    std::cout << endl;

    a = (float *) malloc(sizeof(float) * N);

    // Allocate device memory for a
    hipMalloc((void **) &d_a, sizeof(float) * N);
    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);

    vector_add << < 1, 1 >> > (out, d_a, b, N);

    for (int i = 0; i < N; i++) {
        std::cout << out[i] << " ";
    }
    std::cout << endl;


    // Cleanup after kernel execution
    hipFree(d_a);
    free(a);

    std::cout << "end\n";
    return 0;
}
